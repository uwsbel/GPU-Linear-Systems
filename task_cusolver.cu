#include <iostream>
#include <vector>
#include <fstream>
#include <string>
#include <cmath>
#include <algorithm>
#include <chrono>
#include <tuple>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsparse.h>

// Error checking macro for CUDA API calls
#define CHECK_CUDA(func)                                               \
{                                                                      \
    hipError_t status = (func);                                       \
    if (status != hipSuccess) {                                       \
        printf("CUDA API failed at %s line %d with error: %s (%d)\n",  \
               __FILE__, __LINE__, hipGetErrorString(status), status);\
        exit(EXIT_FAILURE);                                            \
    }                                                                  \
}

// Error checking macro for cuSOLVER API calls
#define CHECK_CUSOLVER(func)                                             \
{                                                                        \
    hipsolverStatus_t status = (func);                                    \
    if (status != HIPSOLVER_STATUS_SUCCESS) {                             \
        printf("cuSOLVER API failed at %s line %d with error: %d\n",     \
               __FILE__, __LINE__, status);                              \
        exit(EXIT_FAILURE);                                              \
    }                                                                    \
}

// Error checking macro for cuSPARSE API calls
#define CHECK_CUSPARSE(func)                                             \
{                                                                        \
    hipsparseStatus_t status = (func);                                    \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                             \
        printf("cuSPARSE API failed at %s line %d with error: %d\n",     \
               __FILE__, __LINE__, status);                              \
        exit(EXIT_FAILURE);                                              \
    }                                                                    \
}

// Function to read matrix from file in COO format and convert to CSR format
void readMatrixCSR(const std::string& filename, 
                  std::vector<double>& values, 
                  std::vector<int>& rowIndex, 
                  std::vector<int>& columns,
                  int& n) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open file " << filename << std::endl;
        exit(1);
    }

    // Read all triplets first to determine matrix size
    std::vector<std::tuple<int, int, double>> triplets;
    int row, col;
    double value;
    int max_row = 0, max_col = 0;
    
    // Read all entries
    while (file >> row >> col >> value) {
        // Convert from 1-based to 0-based indexing if needed
        row--;
        col--;
        
        // Keep track of matrix dimensions
        max_row = std::max(max_row, row);
        max_col = std::max(max_col, col);
        
        triplets.emplace_back(row, col, value);
    }
    
    // Matrix dimensions are max indices + 1 (since we converted to 0-based)
    n = max_row + 1;
    
    // Check if matrix is square
    if (max_row != max_col) {
        std::cerr << "Error: Matrix is not square. Rows: " << max_row + 1 
                  << ", Cols: " << max_col + 1 << std::endl;
        exit(1);
    }
    
    // Sort triplets by row, then by column for CSR format
    std::sort(triplets.begin(), triplets.end());
    
    // Initialize CSR arrays
    values.resize(triplets.size());
    columns.resize(triplets.size());
    rowIndex.resize(n + 1, 0);
    
    // Fill in the CSR arrays
    int current_row = -1;
    for (size_t i = 0; i < triplets.size(); i++) {
        int row = std::get<0>(triplets[i]);
        int col = std::get<1>(triplets[i]);
        double val = std::get<2>(triplets[i]);
        
        // Update row index array
        while (current_row < row) {
            current_row++;
            rowIndex[current_row] = i;
        }
        
        // Store column index and value
        columns[i] = col;
        values[i] = val;
    }
    
    // Set the last element of rowIndex
    rowIndex[n] = triplets.size();
    
    file.close();
}

// Function to read vector from file
std::vector<double> readVector(const std::string& filename) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open file " << filename << std::endl;
        exit(1);
    }

    std::vector<double> values;
    double value;
    
    // Read all values from the file
    while (file >> value) {
        values.push_back(value);
    }
        
    // Check if we read anything
    if (values.empty()) {
        std::cerr << "Warning: No data read from " << filename << std::endl;
    }

    file.close();
    return values;
}

// Function to read the known solution (combining Dl and Dv)
std::vector<double> readKnownSolution(const std::string& dvFilename, const std::string& dlFilename) {
    std::vector<double> dvPart = readVector(dvFilename);
    std::vector<double> dlPart = readVector(dlFilename);
    
    // Negate dlPart before combining
    for (auto& val : dlPart) {
        val = -val;
    }
    
    // Create combined vector
    std::vector<double> solution;
    solution.reserve(dvPart.size() + dlPart.size());
    solution.insert(solution.end(), dvPart.begin(), dvPart.end());
    solution.insert(solution.end(), dlPart.begin(), dlPart.end());
    
    return solution;
}

// Function to write vector to file
void writeVectorToFile(const std::vector<double>& vector, const std::string& filename) {
    std::ofstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open file " << filename << " for writing" << std::endl;
        exit(1);
    }

    // Set precision for output
    file.precision(16);
    file << std::scientific;

    // Write each element on a new line
    for (size_t i = 0; i < vector.size(); i++) {
        file << vector[i] << std::endl;
    }
    
    file.close();
        std::cout << "Solution written to " << filename << std::endl;
}

// Calculate relative error between two vectors
double calculateRelativeError(const std::vector<double>& computed, const std::vector<double>& reference) {
    if (computed.size() != reference.size()) {
        std::cerr << "Error: Vector sizes don't match for error calculation" << std::endl;
        return -1.0;
    }
    
    double norm_diff = 0.0;
    double norm_ref = 0.0;
    
    for (size_t i = 0; i < computed.size(); i++) {
        double diff = computed[i] - reference[i];
        norm_diff += diff * diff;
        norm_ref += reference[i] * reference[i];
    }
    
    return std::sqrt(norm_diff) / std::sqrt(norm_ref);
}

int main(int argc, char* argv[]) {
    // Set CUDA device to 0 (first GPU)
    int deviceId = 0;
    CHECK_CUDA(hipSetDevice(deviceId));
    
    // Print device info
    hipDeviceProp_t prop;
    CHECK_CUDA(hipGetDeviceProperties(&prop, deviceId));
    std::cout << "Using GPU device: " << prop.name << std::endl;
    
    // File paths
    std::string matrixFile = "data/ancf/80/solve_2002_0_Z.dat";
    std::string rhsFile = "data/ancf/80/solve_2002_0_rhs.dat";
    std::string dvFile = "data/ancf/80/solve_2002_0_Dv.dat";
    std::string dlFile = "data/ancf/80/solve_2002_0_Dl.dat";
    std::string outputFile = "soln_cusolver_80.dat";
    
    // Read matrix in CSR format
    std::vector<double> csrValues;
    std::vector<int> csrRowPtr;
    std::vector<int> csrColInd;
    int n;
    
    std::cout << "Reading matrix from " << matrixFile << std::endl;
    readMatrixCSR(matrixFile, csrValues, csrRowPtr, csrColInd, n);
    std::cout << "Matrix size: " << n << "x" << n << " with " << csrValues.size() << " non-zero elements" << std::endl;
    
    // Read right-hand side
    std::cout << "Reading RHS from " << rhsFile << std::endl;
    std::vector<double> rhs = readVector(rhsFile);
    if (rhs.size() != static_cast<size_t>(n)) {
        std::cerr << "Error: RHS vector size (" << rhs.size() 
                  << ") does not match matrix size (" << n << ")" << std::endl;
        exit(1);
    }
    
    // Read known solution (for error calculation)
    std::cout << "Reading known solution" << std::endl;
    std::vector<double> knownSolution = readKnownSolution(dvFile, dlFile);
    
    // Prepare solution vector
    std::vector<double> solution(n, 0.0);
    
    // Create cuSOLVER and cuSPARSE handles
    hipsolverSpHandle_t cusolverHandle = nullptr;
    hipsparseHandle_t cusparseHandle = nullptr;
    CHECK_CUSOLVER(hipsolverSpCreate(&cusolverHandle));
    CHECK_CUSPARSE(hipsparseCreate(&cusparseHandle));
    
    // Allocate device memory
    double *d_csrValues = nullptr;
    int *d_csrRowPtr = nullptr;
    int *d_csrColInd = nullptr;
    double *d_rhs = nullptr;
    double *d_solution = nullptr;
    
    CHECK_CUDA(hipMalloc((void**)&d_csrValues, csrValues.size() * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**)&d_csrRowPtr, (n + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_csrColInd, csrColInd.size() * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_rhs, n * sizeof(double)));
    CHECK_CUDA(hipMalloc((void**)&d_solution, n * sizeof(double)));
    
    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_csrValues, csrValues.data(), csrValues.size() * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrRowPtr, csrRowPtr.data(), (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrColInd, csrColInd.data(), csrColInd.size() * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_rhs, rhs.data(), n * sizeof(double), hipMemcpyHostToDevice));
    
    // Create matrix descriptor
    hipsparseMatDescr_t matDescr = nullptr;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&matDescr));
    CHECK_CUSPARSE(hipsparseSetMatType(matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(matDescr, HIPSPARSE_INDEX_BASE_ZERO));
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    
    // Solve linear system using cuSOLVER
    std::cout << "Solving linear system using cuSOLVER..." << std::endl;
    
    // Start timing
    CHECK_CUDA(hipEventRecord(start));
    
    // Setup for the solver
    int singularity = 0;
    
    // Create parameter structure for the solver
    // Using LU factorization as requested
    
    // LU factorization with partial pivoting (host version)
    // Needs to be updated
    CHECK_CUSOLVER(hipsolverSpDcsrlsvqr(
        cusolverHandle, n, csrValues.size(),
        matDescr, d_csrValues, d_csrRowPtr, d_csrColInd,  // DEVICE pointers
        d_rhs, 1e-12, // tolerance
        1,  // reorder = 1 means use symrcm reordering
        d_solution, &singularity));  // DEVICE solution vector

    // Stop timing
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    
    // Calculate elapsed time
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    
    // Copy solution back to host
    CHECK_CUDA(hipMemcpy(solution.data(), d_solution, n * sizeof(double), hipMemcpyDeviceToHost));
    
    // Check for singularity
    if (singularity >= 0) {
        std::cout << "WARNING: The matrix is singular at row " << singularity << std::endl;
    }
    
    // Calculate relative error
    double relError = calculateRelativeError(solution, knownSolution);
    
    // Output results
    std::cout << "Time to solve: " << milliseconds << " ms" << std::endl;
    std::cout << "Relative error: " << relError << std::endl;
    
    // Write solution to file
    writeVectorToFile(solution, outputFile);
    std::cout << "Solution written to " << outputFile << std::endl;
    
    // Clean up CUDA events
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    
    // Clean up
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(matDescr));
    CHECK_CUSOLVER(hipsolverSpDestroy(cusolverHandle));
    CHECK_CUSPARSE(hipsparseDestroy(cusparseHandle));
    
    CHECK_CUDA(hipFree(d_csrValues));
    CHECK_CUDA(hipFree(d_csrRowPtr));
    CHECK_CUDA(hipFree(d_csrColInd));
    CHECK_CUDA(hipFree(d_rhs));
    CHECK_CUDA(hipFree(d_solution));
    
    return 0;
}
